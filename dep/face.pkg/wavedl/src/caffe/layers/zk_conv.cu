#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/zk_conv.hpp"
#include "" 

#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

namespace caffe {
	template <typename Dtype>
	__global__ void backward_bias_gpuwithgroup(const int Batchsize, 
		const Dtype *top_diff, const int top_rows, const int top_cols, const int top_depth, 
		Dtype *bias_diff,const int bias_Num){
		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
		if (thread_id >= bias_Num) return;
		const int top_diff_offset = thread_id*top_rows*top_cols;
		Dtype gradient = 0;
		for (int n = 0; n < Batchsize; n++)	{
			const int top_offset = n*top_depth*top_cols*top_rows;
			for (int i = 0; i < top_rows*top_cols; i+=4){
				gradient += top_diff[top_offset + top_diff_offset + i];
				gradient += top_diff[top_offset + top_diff_offset + i+1];
				gradient += top_diff[top_offset + top_diff_offset + i+2];
				gradient += top_diff[top_offset + top_diff_offset + i+3];
			}			
		}		
		bias_diff[thread_id] = gradient;
	}

	/*��bottom�ݶȵķ��򴫲�Ϊtop_diff��weight��ת180�Ⱥ���о��
	* �����̼߳���ÿһ��bottom���ݶ�
	*/
	template <typename Dtype>
	__global__ void backward_Input_gpuwithgroup(const Dtype *top_diff, const int top_rows, const int top_cols, const int top_depth, const int top_group,
		const Dtype *weight, const int weight_rows, const int weight_cols, const int weight_depth,
		const int stride, const int pad_rows, const int pad_cols,
		const int bottom_rows, const int bottom_cols, const int bottom_depth, const int bottom_group, Dtype *bottom_diff, const int bottom_Number){

		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
		if (thread_id >= bottom_Number) return;

		const int B_col = thread_id%bottom_cols;
		const int B_row = (thread_id / bottom_cols) % bottom_rows;
		const int B_chanel = (thread_id / bottom_cols / bottom_rows) % bottom_depth;
		const int B_batch = thread_id / bottom_cols / bottom_rows / bottom_depth;

		/* bottom_groupΪ����ÿ����������ͨ����
		*top_groupΪ���ÿ����������ͨ����
		*���㷽ʽ�����ȸ���B_chanel�������ǰͨ�����ڵڼ���Nth_group=B_chanel/bottom_group;
		*����Nth_group��������top_diff��ƫ��top_offset=Nth_group*top_group*top_cols*top_rows;
		*Ȩ��weightƫ��kernel_offset=OD*in_group*kernel_cols*kernel_rows;
		*/

		int Nth_group = B_chanel / bottom_group;
		int Nth_chanel = B_chanel%bottom_group;

		const Dtype *top_diff_slice = top_diff + B_batch*top_depth*top_rows*top_cols + Nth_group*top_group*top_cols*top_rows;
		const int weight_rotate_offset = Nth_group*top_group*weight_depth*weight_cols*weight_rows + (Nth_chanel + 1)*weight_rows*weight_cols - 1;


		int top_C_start = B_col + pad_cols - weight_cols + 1;
		int top_C_end = top_C_start + weight_cols;
		int top_R_start = B_row + pad_rows - weight_rows + 1;
		int top_R_end = top_R_start + weight_rows;

		Dtype gradient = 0;
		for (int top_Cindex = 0; top_Cindex < top_group; top_Cindex++){
			int top_diff_slice_offset = top_Cindex*top_rows*top_cols;
			int weight_offset = top_Cindex*weight_depth*weight_cols*weight_rows+weight_rotate_offset;
			for (int top_R = top_R_start; top_R < top_R_end; top_R++){
				for (int top_C = top_C_start; top_C < top_C_end; top_C++){
					if (top_R%stride == 0 && top_C%stride == 0){
						int top_r = top_R / stride;
						int top_c = top_C / stride;
						if (top_r >= 0 && top_r < top_rows&&top_c >= 0 && top_c < top_cols){
							int topoffset = top_r*top_cols + top_c;
							//std::cout << "gradient=" << thread_id << "  " << top_diff_slice_offset + topoffset << " * " << weight_offset << endl;
							gradient += *(top_diff_slice + top_diff_slice_offset + topoffset)*(*(weight + weight_offset));
						}
					}
					weight_offset--;
				}
			}
		}
		bottom_diff[thread_id]=gradient;
	}
	


	
	///��Ȩ�ص��ݶ�Ϊ input �� output�ľ��
	//���㷽ʽΪ�����߳�ֱ�Ӽ���Ȩ���ݶȵ�ÿһ��ֵ
	/*groupΪ������ʱ*/
	//numΪbatchsize��
	template <typename Dtype>
	__global__ void backward_weight_gpuwithgroup(const int num, const Dtype *inputdata, const int in_group, const int in_rows, const int in_cols, const int in_depth,
		const Dtype *output_diff, const int out_group, const int out_rows, const int out_cols, const int out_depth,
		const int stride, const int pad_rows, const int pad_cols,
		const int weight_rows, const int weight_cols, const int weight_depth, Dtype *weight_diff, const int weight_Number){


		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
		if (thread_id >= weight_Number) return;

		///Ȩ�ص����з�ʽΪNxMxKxK; NΪ���������MΪ�����ͨ������KΪkernel�ĸ߶ȺͿ��
		const int K_col = thread_id % weight_cols;													//Kernel����
		const int K_row = (thread_id / weight_cols) % weight_rows;									//Kernel����
		const int K_chanel = (thread_id / weight_cols / weight_rows) % weight_depth;				//Kernel��ͨ�����������ͨ����
		const int K_N = thread_id / weight_cols / weight_rows / weight_depth;						//Kernel�ĸ��� �������ͨ����


		/* in_groupΪ����ÿ����������ͨ����
		*out_groupΪ���ÿ����������ͨ����
		*���㷽ʽ�����ȸ���OD�������ǰͨ�����ڵڼ���Nth_group=OD/out_group;
		*����Nth_group��������inputdata��ƫ��in_offset=Nth_group*in_group*in_cols*in_rows;
		*Ȩ��weightƫ��kernel_offset=OD*in_group*kernel_cols*kernel_rows;
		*/
		const int Nth_group = K_N / out_group;

		const int input_offset = in_rows*in_cols*in_depth;
		const int out_offset = out_rows*out_cols*out_depth;
		//const int input_row_start = K_row*stride - pad_rows;
		Dtype gradient = 0;
		for (int n = 0; n < num; n++){
			const Dtype* const inputdata_slice = inputdata + n*input_offset + K_chanel*in_cols*in_rows + Nth_group*in_group*in_cols*in_rows;					//nΪbatsize�еĵڼ��� ����ƫ��
			const Dtype* const outputdata_slice = output_diff + n*out_offset + K_N*out_cols*out_rows;					//���ƫ��
			for (int out_R = 0; out_R < out_rows; out_R++){
				const int in_R = out_R*stride + K_row - pad_rows;
				if (in_R >= 0 && in_R<in_rows){
					for (int out_C = 0; out_C < out_cols; out_C++){
						const int in_C = out_C*stride + K_col - pad_cols;
						if (in_C >= 0 && in_C<in_cols){
							int in_index = in_R*in_cols + in_C;
							int out_index = out_R*out_cols + out_C;
							gradient += inputdata_slice[in_index] * outputdata_slice[out_index];
						}
					}
				}
			}			
		}
		weight_diff[thread_id] = gradient;

	}
	



	
	/*�����̼߳���������ͨ��im2col��ʽ��������Ŀǰ�汾������group��ʽ
	*���巽ʽ������ʾ����Ҫ����Ĳ���
	* inputdata		     	��������
	* in_group				������ÿ����������ͨ����
	* in_rows				����ͼ��߶�
	* in_cols				����ͼ����
	* in_depth				����ͼ����ͨ����
	* weight                ���Ȩ��
	* weight_rows           ����˵ĸ߶�
	* weight_cols			����˵Ŀ��
	* stride                ����
	* pad_rows              pad��
	* pad_cols              pad��
	* biasexist             bias�Ƿ���ڵı�־
	* biasdata              bias����
	* out_group			 	���ͼ����ÿ����������ͨ����
	* out_rows				���ͼ��߶�
	* out_cols				���ͼ����
	* out_depth             ���ͼ����ͨ����
	* outputdata            ������ݣ�
	* out_Number            ��������ܳ���
	*/
	template <typename Dtype>
	__global__ void forward_convwithgroup(const Dtype *inputdata, const int in_group, const int in_rows, const int in_cols, const int in_depth,
		const Dtype *weight, const int weight_rows, const int weight_cols, const int stride, const int pad_rows, const int pad_cols, bool biasexist, const Dtype *biasdata,
		const int out_group, const int out_rows, const int out_cols, const int out_depth, Dtype *outputdata, const int out_Number){
		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
		if (thread_id >= out_Number) return;
		const int OC = thread_id % out_cols;//width
		const int OR = (thread_id / out_cols) % out_rows;//height
		const int OD = (thread_id / out_cols / out_rows) % out_depth;//channel
		const int OB = thread_id / out_cols / out_rows / out_depth;//batch size

		const int input_row_start = OR * stride - pad_rows;
		const int input_col_start = OC * stride - pad_cols;
		const int input_row_end = input_row_start + weight_rows;
		const int input_col_end = input_col_start + weight_cols;

		/* in_groupΪ����ÿ����������ͨ����
		*out_groupΪ���ÿ����������ͨ����
		*���㷽ʽ�����ȸ���OD�������ǰͨ�����ڵڼ���Nth_group=OD/out_group;
		*����Nth_group��������inputdata��ƫ��in_offset=Nth_group*in_group*in_cols*in_rows;
		*Ȩ��weightƫ��kernel_offset=OD*in_group*kernel_cols*kernel_rows;
		*/
		int Nth_group = OD / out_group;
		const int inputoffset = OB*in_depth*in_rows*in_cols + Nth_group*in_group*in_cols*in_rows; // +OD*in_cols*in_rows;// +input_row_start*in_cols + input_col_start;
		const int kerneloffset = OD*in_group*weight_cols*weight_rows;

		Dtype sum = 0.f;
		//#pragma unroll
		for (int W_C = 0; W_C < in_group; W_C++){
			int InputOffset = inputoffset + W_C*in_rows*in_cols;
			int KernelOffset = kerneloffset + W_C*weight_cols*weight_rows;
			//#pragma unroll
			for (int W_H = 0; W_H < weight_rows; W_H++){

				int in_r = input_row_start + W_H;
				//#pragma unroll
				for (int W_W = 0; W_W < weight_cols; W_W++){
					int in_c = input_col_start + W_W;
					if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols) {
						int input_offset = InputOffset + in_r*in_cols + in_c;
						int weight_offset = KernelOffset + W_H*weight_cols + W_W;
						sum += (*(inputdata + input_offset))*(*(weight + weight_offset));
					}

				}
			}
		}
		if (biasexist) outputdata[thread_id] = sum + biasdata[OD];
		else outputdata[thread_id] = sum;
	}
	
	
	
//	template <>
//	void gpu_convwithgroup<float>(const float *inputdata, const int in_group, const int in_rows, const int in_cols, const int in_depth,
//		const float *weight, const int weight_rows, const int weight_cols, 
//		const int stride, const int pad_rows, const int pad_cols, 
//		bool biasexist, const float *biasdata,
//		const int out_group, const int out_rows, const int out_cols, const int out_depth, float *outputdata, const int out_Number){
//		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
//		if (thread_id >= out_Number) return;
//		const int OC = thread_id % out_cols;//width
//		const int OR = (thread_id / out_cols) % out_rows;//height
//		const int OD = (thread_id / out_cols / out_rows) % out_depth;//channel
//		const int OB = thread_id / out_cols / out_rows / out_depth;//batch size
//
//		const int input_row_start = OR * stride - pad_rows;
//		const int input_col_start = OC * stride - pad_cols;
//		const int input_row_end = input_row_start + weight_rows;
//		const int input_col_end = input_col_start + weight_cols;
//
//		/* in_groupΪ����ÿ����������ͨ����
//		*out_groupΪ���ÿ����������ͨ����
//		*���㷽ʽ�����ȸ���OD�������ǰͨ�����ڵڼ���Nth_group=OD/out_group;
//		*����Nth_group��������inputdata��ƫ��in_offset=Nth_group*in_group*in_cols*in_rows;
//		*Ȩ��weightƫ��kernel_offset=OD*in_group*kernel_cols*kernel_rows;
//		*/
//		int Nth_group = OD / out_group;
//		const int inputoffset = OB*in_depth*in_rows*in_cols + Nth_group*in_group*in_cols*in_rows; // +OD*in_cols*in_rows;// +input_row_start*in_cols + input_col_start;
//		const int kerneloffset = OD*in_group*weight_cols*weight_rows;
//
//		float sum = 0.f;
////#pragma unroll
//		for (int W_C = 0; W_C < in_group; W_C++){
//			int InputOffset = inputoffset + W_C*in_rows*in_cols;
//			int KernelOffset = kerneloffset + W_C*weight_cols*weight_rows;
////#pragma unroll
//			for (int W_H = 0; W_H < weight_rows; W_H++){
//
//				int in_r = input_row_start + W_H;
////#pragma unroll
//				for (int W_W = 0; W_W < weight_cols; W_W++){
//					int in_c = input_col_start + W_W;
//					if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols) {
//						int input_offset = InputOffset + in_r*in_cols + in_c;
//						int weight_offset = KernelOffset + W_H*weight_cols + W_W;
//						sum += (*(inputdata + input_offset))*(*(weight + weight_offset));
//					}
//
//				}
//			}
//		}
//		if (biasexist) outputdata[thread_id] = sum + biasdata[OD];
//		else outputdata[thread_id] = sum;
//	}
//	template <>
//	void gpu_convwithgroup<double>(const double *inputdata, const int in_group, const int in_rows, const int in_cols, const int in_depth,
//		const double *weight, const int weight_rows, const int weight_cols, const int stride, const int pad_rows, const int pad_cols, bool biasexist, const double *biasdata,
//		const int out_group, const int out_rows, const int out_cols, const int out_depth, double *outputdata, const int out_Number){
//		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
//		if (thread_id >= out_Number) return;
//		const int OC = thread_id % out_cols;//width
//		const int OR = (thread_id / out_cols) % out_rows;//height
//		const int OD = (thread_id / out_cols / out_rows) % out_depth;//channel
//		const int OB = thread_id / out_cols / out_rows / out_depth;//batch size
//
//		const int input_row_start = OR * stride - pad_rows;
//		const int input_col_start = OC * stride - pad_cols;
//		const int input_row_end = input_row_start + weight_rows;
//		const int input_col_end = input_col_start + weight_cols;
//
//		/* in_groupΪ����ÿ����������ͨ����
//		*out_groupΪ���ÿ����������ͨ����
//		*���㷽ʽ�����ȸ���OD�������ǰͨ�����ڵڼ���Nth_group=OD/out_group;
//		*����Nth_group��������inputdata��ƫ��in_offset=Nth_group*in_group*in_cols*in_rows;
//		*Ȩ��weightƫ��kernel_offset=OD*in_group*kernel_cols*kernel_rows;
//		*/
//		int Nth_group = OD / out_group;
//		const int inputoffset = OB*in_depth*in_rows*in_cols + Nth_group*in_group*in_cols*in_rows; // +OD*in_cols*in_rows;// +input_row_start*in_cols + input_col_start;
//		const int kerneloffset = OD*in_group*weight_cols*weight_rows;
//
//		double sum = 0.f;
////#pragma unroll
//		for (int W_C = 0; W_C < in_group; W_C++){
//			int InputOffset = inputoffset + W_C*in_rows*in_cols;
//			int KernelOffset = kerneloffset + W_C*weight_cols*weight_rows;
////#pragma unroll
//			for (int W_H = 0; W_H < weight_rows; W_H++){
//
//				int in_r = input_row_start + W_H;
////#pragma unroll
//				for (int W_W = 0; W_W < weight_cols; W_W++){
//					int in_c = input_col_start + W_W;
//					if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols) {
//						int input_offset = InputOffset + in_r*in_cols + in_c;
//						int weight_offset = KernelOffset + W_H*weight_cols + W_W;
//						sum += (*(inputdata + input_offset))*(*(weight + weight_offset));
//					}
//
//				}
//			}
//		}
//		if (biasexist) outputdata[thread_id] = sum + biasdata[OD];
//		else outputdata[thread_id] = sum;
//	};




	template <typename Dtype>
	void ZKConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const int group = this->group_;
		const Dtype* weight = this->blobs_[0]->gpu_data();
		const int weight_rows = this->blobs_[0]->height();
		const int weight_cols = this->blobs_[0]->width();

		bool biasexist = this->bias_term_;
		const Dtype *biasdata = NULL;
		if (biasexist) biasdata = this->blobs_[1]->gpu_data();


		int* stride_data = this->stride_.mutable_cpu_data();
		const int stride = stride_data[0];
		
		int *pad_data = this->pad_.mutable_cpu_data();
		const int pad_rows = pad_data[0];
		const int pad_cols = pad_data[1];

		for (int i = 0; i < bottom.size(); ++i) {
			const Dtype* bottom_data = bottom[i]->gpu_data();   //��������
			vector<int> in_shape_ = bottom[i]->shape();			
			const int in_channels_ = in_shape_[1];               //�������ݵ�ͨ����
			const int in_group = in_channels_ / group;        //����ÿ�������ͨ����
			const int in_height_ = in_shape_[2];				  //�������ݵĸ߶�	
			const int in_width_ = in_shape_[3];				  //�������ݵĿ��

			Dtype* top_data = top[i]->mutable_gpu_data();     // �������
			const int count = top[i]->count();				  // ������ݵ��ܳ���
			vector<int> outshape_ = top[i]->shape();         
			const int outchannels_ = outshape_[1];			  //������ݵ�ͨ����
			const int out_group = outchannels_ / group;		  //���ÿ�������ͨ����
			const int outheight_ = outshape_[2];			  //������ݵĸ߶�
			const int outwidth_ = outshape_[3];				  //������ݵĿ��
			
			forward_convwithgroup<Dtype> << <cuda_gridsize(count), CAFFE_CUDA_NUM_THREADS >> > (bottom_data, in_group, in_height_,
				in_width_, in_channels_, weight, weight_rows, weight_cols, stride, pad_rows, pad_cols, biasexist, biasdata,
				out_group, outheight_, outwidth_, outchannels_, top_data, count);
		}
		
	}

	template <typename Dtype>
	void ZKConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		const int batchSize = this->num_;
		const Dtype* weight = this->blobs_[0]->gpu_data();
		Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
		const int weight_num = this->blobs_[0]->count();
		const int weight_rows = this->blobs_[0]->height();
		const int weight_cols = this->blobs_[0]->width();
		const int weight_depth = this->blobs_[0]->channels();

		int* stride_data = this->stride_.mutable_cpu_data();
		const int stride = stride_data[0];

		int *pad_data = this->pad_.mutable_cpu_data();
		const int pad_rows = pad_data[0];
		const int pad_cols = pad_data[1];

		/*const int top_rows, const int top_cols, const int top_depth,
			Dtype *bias_diff, const int bias_Num*/
		//Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
		for (int i = 0; i < top.size(); ++i) {
			const Dtype* top_diff = top[i]->gpu_diff();
			std::vector<int> outshape_ = top[i]->shape();
			const int bias_number = this->conv_out_channels_;
			const int outchannels_ = outshape_[1];			  //������ݵ�ͨ����
			const int out_group = outchannels_ / group_;		  //���ÿ�������ͨ����
			const int outheight_ = outshape_[2];			  //������ݵĸ߶�
			const int outwidth_ = outshape_[3];				  //������ݵĿ��
			// Bias gradient, if necessary.
			if (this->bias_term_ && this->param_propagate_down_[1]){
				Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();				

				backward_bias_gpuwithgroup<Dtype> << <cuda_gridsize(bias_number), CAFFE_CUDA_NUM_THREADS >> >(batchSize, top_diff, outheight_,
					outwidth_, outchannels_, bias_diff, bias_number);
			}

			if (this->param_propagate_down_[0] || propagate_down[i]){
				const Dtype* bottom_data = bottom[i]->gpu_data();
				Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
				const int bottom_number = bottom[i]->count();
				vector<int> in_shape_ = bottom[i]->shape();
				const int in_channels_ = in_shape_[1];               //�������ݵ�ͨ����
				const int in_group = in_channels_ / group_;        //����ÿ�������ͨ����
				const int in_height_ = in_shape_[2];				  //�������ݵĸ߶�	
				const int in_width_ = in_shape_[3];				  //�������ݵĿ��
				
				if (this->param_propagate_down_[0]){

					/*backward_weight_gpuwithgroup(const int num, const Dtype *inputdata, const int in_group, const int in_rows, const int in_cols, const int in_depth,
						const Dtype *output_diff, const int out_group, const int out_rows, const int out_cols, const int out_depth,
						const int stride, const int pad_rows, const int pad_cols,
						const int weight_rows, const int weight_cols, const int weight_depth, Dtype *weight_diff, const int weight_Number)*/
					backward_weight_gpuwithgroup<Dtype> << < cuda_gridsize(weight_num), CAFFE_CUDA_NUM_THREADS >> > (batchSize,
						bottom_data, in_group, in_height_, in_width_, in_channels_,
						top_diff, out_group, outheight_, outwidth_, outchannels_,
						stride, pad_rows, pad_cols,
						weight_rows, weight_cols, weight_depth, weight_diff, weight_num
						);
				}				
				/*backward_Input_gpuwithgroup(const Dtype *top_diff, const int top_rows, const int top_cols, const int top_depth, const int top_group,
				const Dtype *weight, const int weight_rows, const int weight_cols, const int weight_depth,
				const int stride, const int pad_rows, const int pad_cols,
				const int bottom_rows, const int bottom_cols, const int bottom_depth, const int bottom_group, Dtype *bottom_diff, const int bottom_Number)*/
				if (propagate_down[i]) {
					backward_Input_gpuwithgroup<Dtype> << <cuda_gridsize(bottom_number), CAFFE_CUDA_NUM_THREADS >> >(top_diff, outheight_, outwidth_,
						outchannels_, out_group, weight, weight_rows, weight_cols, weight_depth, stride, pad_rows, pad_rows,
						in_height_, in_width_, in_channels_, in_group, bottom_diff, bottom_number);
				}
			}

		}

	}


	//template <typename Dtype>
	//void ZKConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	//	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	//	const Dtype* weight = this->blobs_[0]->gpu_data();
	//	Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
	//	for (int i = 0; i < top.size(); ++i) {
	//		const Dtype* top_diff = top[i]->gpu_diff();
	//		// Bias gradient, if necessary.
	//		if (this->bias_term_ && this->param_propagate_down_[1]) {
	//			Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
	//			for (int n = 0; n < this->num_; ++n) {
	//				this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
	//			}
	//		}
	//		if (this->param_propagate_down_[0] || propagate_down[i]) {
	//			const Dtype* bottom_data = bottom[i]->gpu_data();
	//			Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
	//			for (int n = 0; n < this->num_; ++n) {
	//				// gradient w.r.t. weight. Note that we will accumulate diffs.
	//				if (this->param_propagate_down_[0]) {
	//					this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
	//						top_diff + n * this->top_dim_, weight_diff);
	//				}
	//				// gradient w.r.t. bottom data, if necessary.
	//				if (propagate_down[i]) {
	//					this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
	//						bottom_diff + n * this->bottom_dim_);
	//				}
	//			}
	//		}
	//	}
	//}



	INSTANTIATE_LAYER_GPU_FUNCS(ZKConvolutionLayer);
}